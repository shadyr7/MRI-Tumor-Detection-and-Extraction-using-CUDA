#include "hip/hip_runtime.h"
// PASTE THE *MODIFIED* morphological_ops.cu CODE HERE
#include <stdexcept> // Required for std::runtime_error
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <string>

using namespace cv;

// CUDA error-checking macro
#define CHECK_CUDA(call) do {                                  \
    hipError_t err = call;                                    \
    if (err != hipSuccess) {                                  \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n",          \
               __FILE__, __LINE__, hipGetErrorString(err));   \
        exit(EXIT_FAILURE);                                    \
    }                                                          \
} while(0)

// CUDA Kernel for Thresholding (Binary)
__global__ void thresholdKernel(const unsigned char *input, unsigned char *output, int width, int height, unsigned char threshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        output[idx] = (input[idx] > threshold) ? 255 : 0;
    }
}

// CUDA Kernel for Dilation with 5x5 structuring element
__global__ void dilationKernel(const unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        unsigned char maxVal = 0; // Dilate looks for max in neighborhood

        // Iterate over the 5x5 neighborhood
        for (int ky = -2; ky <= 2; ky++) {
            for (int kx = -2; kx <= 2; kx++) {
                int neighbor_x = x + kx;
                int neighbor_y = y + ky;

                // Check boundaries of the neighbor
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    unsigned char neighbor_val = input[neighbor_y * width + neighbor_x];
                    // Update maxVal if current neighbor is greater
                    if (neighbor_val > maxVal) {
                        maxVal = neighbor_val;
                    }
                    // Optimization: if maxVal is already 255, we can stop searching neighborhood
                    if (maxVal == 255) goto end_dilation_loop;
                }
            }
        }
        end_dilation_loop:; // Label for goto jump

        output[idx] = maxVal;
    }
}

// CUDA Kernel for Erosion with 5x5 structuring element
__global__ void erosionKernel(const unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        unsigned char minVal = 255; // Erode looks for min in neighborhood

        // Iterate over the 5x5 neighborhood
        for (int ky = -2; ky <= 2; ky++) {
            for (int kx = -2; kx <= 2; kx++) {
                int neighbor_x = x + kx;
                int neighbor_y = y + ky;

                 // Check boundaries of the neighbor
                if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                    unsigned char neighbor_val = input[neighbor_y * width + neighbor_x];
                     // Update minVal if current neighbor is smaller
                    if (neighbor_val < minVal) {
                        minVal = neighbor_val;
                    }
                     // Optimization: if minVal is already 0, we can stop searching neighborhood
                     if (minVal == 0) goto end_erosion_loop;
                } else {
                    // Handle border cases for erosion: if neighborhood goes outside,
                    // consider it 0 (minimum possible value) if we are eroding foreground (255)
                    // If we assume the structuring element must be fully contained,
                    // this effectively shrinks the image border.
                    // For simplicity here, we often assume border replication or ignore,
                    // but erosion technically requires minimum, so hitting border implies 0.
                     minVal = 0; // If any part of SE is outside, result is min (0)
                     goto end_erosion_loop;
                }
            }
        }
       end_erosion_loop:; // Label for goto jump

        output[idx] = minVal;
    }
}


// PASTE THIS *ENTIRE main function* into the morphological_ops.cu cell,
// replacing the old main function.

// PASTE THIS *ENTIRE main function* into the morphological_ops.cu cell,
// replacing the old main function.

// PASTE THIS *ENTIRE main function* into the morphological_ops.cu cell,
// replacing the old main function.

int main(int argc, char **argv) {
    // --- Argument Parsing & Initial Checks ---
    if (argc != 6) {
        fprintf(stderr, "Usage: %s <input_edge_image> <output_refined_image> <threshold_val> <dbg_binary_out> <dbg_dilated_out>\n", argv[0]);
        fprintf(stderr, "  threshold_val: Value (0-255) for initial binarization.\n");
        fprintf(stderr, "  dbg_binary_out: Filename to save intermediate binary image.\n");
        fprintf(stderr, "  dbg_dilated_out: Filename to save intermediate dilated image.\n");
        return -1;
    }
    const char* inputFilename = argv[1];
    const char* outputFilename = argv[2];
    int threshold_val = atoi(argv[3]);
    const char* binaryOutputFilename = argv[4];
    const char* dilatedOutputFilename = argv[5];

    if (threshold_val < 0 || threshold_val > 255) {
        fprintf(stderr, "Error: Threshold value must be between 0 and 255. Got %d\n", threshold_val);
        return -1;
    }

    // --- Variable Declarations (Initialize where possible) ---
    hipEvent_t start = nullptr, stop = nullptr;
    float elapsedTime = 0.0f;
    hipError_t err = hipSuccess; // Track status
    Mat edgeImage;
    Mat binaryImage, dilatedImage, refinedEdges;
    int width = 0;
    int height = 0;
    size_t image_size = 0;
    unsigned char *h_edges = nullptr;
    unsigned char *h_binary = nullptr;
    unsigned char *h_dilated = nullptr;
    unsigned char *h_final = nullptr;
    unsigned char *d_edges = nullptr, *d_binary = nullptr, *d_dilated = nullptr, *d_final = nullptr;
    dim3 blockSize(16, 16);
    dim3 gridSize;

    // --- Resource Acquisition & Processing Block ---

    // Create Events
    err = hipEventCreate(&start); if (err != hipSuccess) { fprintf(stderr, "hipEventCreate failed for start: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipEventCreate(&stop); if (err != hipSuccess) { fprintf(stderr, "hipEventCreate failed for stop: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipEventRecord(start, 0); if (err != hipSuccess) { fprintf(stderr, "hipEventRecord failed for start: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Load Image
    edgeImage = imread(inputFilename, IMREAD_GRAYSCALE);
    if (edgeImage.empty()) {
        fprintf(stderr, "Error: Edge-detected image not found or could not load: %s\n", inputFilename);
        err = hipErrorFileNotFound; // Indicate error type
        goto cleanup_and_exit;
    }
    if (edgeImage.type() != CV_8UC1) {
        fprintf(stderr, "Error: Input image must be 8-bit 1-channel grayscale (CV_8UC1). Found type %d\n", edgeImage.type());
        err = hipErrorInvalidValue;
        goto cleanup_and_exit;
    }

    // Get dimensions AFTER successful load
    width = edgeImage.cols;
    height = edgeImage.rows;
    image_size = (size_t)width * height * sizeof(unsigned char);
    printf("Morphological Ops Input: %s (%dx%d), Threshold: %d\n", inputFilename, width, height, threshold_val);

    // Handle image continuity
    h_edges = edgeImage.data;
    if (!edgeImage.isContinuous()) {
        fprintf(stderr, "Warning: Input edge image data is not continuous. Cloning.\n");
        edgeImage = edgeImage.clone();
        h_edges = edgeImage.data;
    }

    // Allocate Host Memory
    h_binary = (unsigned char*)malloc(image_size);
    h_dilated = (unsigned char*)malloc(image_size);
    h_final = (unsigned char*)malloc(image_size);
    if (!h_binary || !h_dilated || !h_final) {
        fprintf(stderr, "Error: Unable to allocate host memory for outputs!\n");
        err = hipErrorOutOfMemory;
        goto cleanup_and_exit;
    }

    // Allocate Device Memory
    err = hipMalloc(&d_edges, image_size); if (err != hipSuccess) { fprintf(stderr, "hipMalloc failed for d_edges: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipMalloc(&d_binary, image_size); if (err != hipSuccess) { fprintf(stderr, "hipMalloc failed for d_binary: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipMalloc(&d_dilated, image_size); if (err != hipSuccess) { fprintf(stderr, "hipMalloc failed for d_dilated: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipMalloc(&d_final, image_size); if (err != hipSuccess) { fprintf(stderr, "hipMalloc failed for d_final: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Copy input to device
    err = hipMemcpy(d_edges, h_edges, image_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "hipMemcpy H2D failed: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Calculate Grid Size
    gridSize = dim3((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // --- CUDA Kernels & Intermediate Steps ---

    // 1. Thresholding
    thresholdKernel<<<gridSize, blockSize>>>(d_edges, d_binary, width, height, (unsigned char)threshold_val);
    err = hipGetLastError(); // Check after kernel
    if (err != hipSuccess) { fprintf(stderr, "CUDA Kernel Error (thresholdKernel): %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipDeviceSynchronize(); // Check after sync
    if (err != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after threshold failed: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Save intermediate binary
    err = hipMemcpy(h_binary, d_binary, image_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "hipMemcpy D2H failed for binary: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    binaryImage = Mat(height, width, CV_8UC1, h_binary);
    if (!imwrite(binaryOutputFilename, binaryImage)) { fprintf(stderr, "Warning: Could not save intermediate binary image %s\n", binaryOutputFilename); }
    else { printf("Morphological Ops Debug: Saved binary image %s\n", binaryOutputFilename); }


    // 2. Dilation
    dilationKernel<<<gridSize, blockSize>>>(d_binary, d_dilated, width, height);
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "CUDA Kernel Error (dilationKernel): %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipDeviceSynchronize();
     if (err != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after dilation failed: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Save intermediate dilated
    err = hipMemcpy(h_dilated, d_dilated, image_size, hipMemcpyDeviceToHost);
     if (err != hipSuccess) { fprintf(stderr, "hipMemcpy D2H failed for dilated: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    dilatedImage = Mat(height, width, CV_8UC1, h_dilated);
    if (!imwrite(dilatedOutputFilename, dilatedImage)) { fprintf(stderr, "Warning: Could not save intermediate dilated image %s\n", dilatedOutputFilename); }
    else { printf("Morphological Ops Debug: Saved dilated image %s\n", dilatedOutputFilename); }


    // 3. Erosion
    erosionKernel<<<gridSize, blockSize>>>(d_dilated, d_final, width, height);
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "CUDA Kernel Error (erosionKernel): %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipDeviceSynchronize();
     if (err != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after erosion failed: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // --- Timing & Final Output ---
    // Only time if everything up to here succeeded
    err = hipEventRecord(stop, 0); if (err != hipSuccess) { fprintf(stderr, "hipEventRecord failed for stop: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipEventSynchronize(stop); if (err != hipSuccess) { fprintf(stderr, "hipEventSynchronize failed for stop: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }
    err = hipEventElapsedTime(&elapsedTime, start, stop); if (err != hipSuccess) { fprintf(stderr, "hipEventElapsedTime failed: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Copy final result back
    err = hipMemcpy(h_final, d_final, image_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "hipMemcpy D2H failed for final: %s\n", hipGetErrorString(err)); goto cleanup_and_exit; }

    // Save final image
    refinedEdges = Mat(height, width, CV_8UC1, h_final);
    if (!imwrite(outputFilename, refinedEdges)) { fprintf(stderr, "Error: Could not save refined edges image %s\n", outputFilename); /* Potentially non-fatal? */ }
    else { printf("Morphological Ops Output: %s\n", outputFilename); }

cleanup_and_exit:
    // --- Unified Cleanup ---
    if (d_edges) hipFree(d_edges);
    if (d_binary) hipFree(d_binary);
    if (d_dilated) hipFree(d_dilated);
    if (d_final) hipFree(d_final);

    free(h_binary);
    free(h_dilated);
    free(h_final);

    if(start) hipEventDestroy(start);
    if(stop) hipEventDestroy(stop);

    if (err == hipSuccess && elapsedTime > 0) {
        printf("Morphological Ops Time: %.3f ms\n", elapsedTime);
    } else if (err != hipSuccess) {
         printf("Morphological Ops finished with error: %s\n", hipGetErrorString(err));
    }

    return (err == hipSuccess) ? 0 : -1;
}
